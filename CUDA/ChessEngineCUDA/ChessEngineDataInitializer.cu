
#include <sstream>

#include "ExceptionsMacro.h"

#include "Logger.h"
#include "ChessEngineConfigurationFileReaderWriter.h"
#include "ChessEngineAllPossibleMovesGenerator.h"

#include "ChessEngineDataInitializer.h"

using namespace std;

void ChessEngineDataInitializer::ClearDataForAllPossibleMovesGeneratorAlgorithm()
{
	try
	{
		ChessEngineAllPossibleMovesGeneratorLocalReference.DeepLevel = 0;
		ChessEngineAllPossibleMovesGeneratorLocalReference.IsAnyPawnPromoted = false;

		for (ChessMove& InvestigatedMove : ChessEngineAllPossibleMovesGeneratorLocalReference.ActuallyInvestigatedMovesPath)	
			InvestigatedMove = ChessMove{ 0, 0, 0, 0, 0, 0, 0 };
	}
	CATCH("clearing chess analzyer algorithm data");
}

void ChessEngineDataInitializer::ClearPiecesData()
{
	try
	{
		for (ChessPiece& Piece : ChessEngineAllPossibleMovesGeneratorLocalReference.Pieces)
		{
			Piece.ActualPosX = 0;
			Piece.ActualPosY = 0;
			Piece.Alive = false;
			Piece.Promoted = false;
		}
	}
	CATCH("clearing pieces data");
}

void ChessEngineDataInitializer::SetPieceStartProperData(const PosType CX, const PosType CY, const PieceNumType PieceNum)
{
	try
	{
		ChessEngineAllPossibleMovesGeneratorLocalReference.Pieces[PieceNum].ActualPosX = CX;
		ChessEngineAllPossibleMovesGeneratorLocalReference.Pieces[PieceNum].ActualPosY = CY;
		ChessEngineAllPossibleMovesGeneratorLocalReference.Pieces[PieceNum].Alive = true;

		ChessEngineAllPossibleMovesGeneratorLocalReference.ChessBoard[CX][CY] = PieceNum;

		if ((ChessEngineAllPossibleMovesGeneratorLocalReference.IsPawn<White>(PieceNum) == true && CY == 1) || (ChessEngineAllPossibleMovesGeneratorLocalReference.IsPawn<Black>(PieceNum) == true && CY == MaxChessSizeY - 1))
		{
			ChessEngineAllPossibleMovesGeneratorLocalReference.Pieces[PieceNum].Promoted = true;
			ChessEngineAllPossibleMovesGeneratorLocalReference.IsAnyPawnPromoted = true;
		}
	}
	CATCH("set piece start proper data")
}

void ChessEngineDataInitializer::SetStartPromotedPawn(const PosType CX, const PosType CY, const PieceNumType PromotedPawnPieceNum, const PieceNumType PromotedPawnNum)
{
	try
	{
		if (ChessEngineAllPossibleMovesGeneratorLocalReference.Pieces[PromotedPawnPieceNum].Alive == false)
		{
			SetPieceStartProperData(CX, CY, PromotedPawnPieceNum);
			ChessEngineAllPossibleMovesGeneratorLocalReference.Pieces[PromotedPawnPieceNum].Promoted = true;
			ChessEngineAllPossibleMovesGeneratorLocalReference.IsAnyPawnPromoted = true;
		}
		else
			throw runtime_error(string(string("Pawn ") + PromotedPawnString[PromotedPawnNum] + " could not be promoted because already exists on chessboard as normal pawn").c_str());
	}
	CATCH_AND_THROW("setting start promoted pawn")
}

bool ChessEngineDataInitializer::FindChessMainPiece(const PosType CX, const PosType CY)
{
	try
	{
		if (ChessEngineAllPossibleMovesGeneratorLocalReference.ChessBoardStartData[CX][CY] != EmptyFieldString)
			for (PieceNumType PieceNum = 0; PieceNum < NumberOfPieces; PieceNum++)
				if (ChessEngineAllPossibleMovesGeneratorLocalReference.ChessBoardStartData[CX][CY] == ChessEngineConfigurationFileReaderWriterObject.PieceString[PieceNum])
				{
					PieceNumType PieceNumForChessBoard;
					if (ChessEngineConfigurationFileReaderWriterObject.ReverseColorOfPieces == true)
						PieceNumForChessBoard = (PieceNum <= EndOfWhitePiecesNum ? PieceNum + EndOfWhitePiecesNum + 1 : PieceNum - EndOfWhitePiecesNum - 1);
					else
						PieceNumForChessBoard = PieceNum;

					if (ChessEngineAllPossibleMovesGeneratorLocalReference.Pieces[PieceNumForChessBoard].Alive == false)
					{
						if ((ChessEngineAllPossibleMovesGeneratorLocalReference.IsPawn<White>(PieceNumForChessBoard) == true && CY == MaxChessSizeY - 1) || (ChessEngineAllPossibleMovesGeneratorLocalReference.IsPawn<Black>(PieceNumForChessBoard) == true && CY == 1))
							throw runtime_error(string(string("Pawn ") + ChessEngineConfigurationFileReaderWriterObject.PieceString[PieceNumForChessBoard] + " in impossible start position").c_str());

						SetPieceStartProperData(CX, CY, PieceNumForChessBoard);

						return true;
					}
					else
						throw runtime_error(string(string("Piece ") + ChessEngineConfigurationFileReaderWriterObject.PieceString[PieceNumForChessBoard] + " already exists on chessboard").c_str());
				}
	}
	CATCH_AND_THROW("finding chess main piece")

	return false;
}

bool ChessEngineDataInitializer::FindChessPromotedPiece(const PosType CX, const PosType CY, const bool FoundChessMainPiece)
{
	try
	{
		if (ChessEngineAllPossibleMovesGeneratorLocalReference.ChessBoardStartData[CX][CY] != EmptyFieldString && FoundChessMainPiece == false)
			for (PieceNumType PromotedPawnNum = 0; PromotedPawnNum < MaxNumberOfPromotedPawns; PromotedPawnNum++)
				if (ChessEngineAllPossibleMovesGeneratorLocalReference.ChessBoardStartData[CX][CY] == PromotedPawnString[PromotedPawnNum])
				{
					if (ChessEngineConfigurationFileReaderWriterObject.ReverseColorOfPieces == false)
					{
						if (PromotedPawnNum < 8)
							SetStartPromotedPawn(CX, CY, Pawn1Num[White] + PromotedPawnNum, PromotedPawnNum);
						else
						if (PromotedPawnNum >= 8)
							SetStartPromotedPawn(CX, CY, Pawn1Num[Black] + PromotedPawnNum - 8, PromotedPawnNum);
					}
					else
					{
						if (PromotedPawnNum < 8)
							SetStartPromotedPawn(CX, CY, Pawn1Num[Black] + PromotedPawnNum, PromotedPawnNum);
						else
						if (PromotedPawnNum >= 8)
							SetStartPromotedPawn(CX, CY, Pawn1Num[White] + PromotedPawnNum - 8, PromotedPawnNum);
					}
					return true;
				}
	}
	CATCH_AND_THROW("finding chess promoted piece")

	return false;
}

void ChessEngineDataInitializer::FindIfThereAreKingsOnChessBoard() const
{
	try
	{
		if (ChessEngineAllPossibleMovesGeneratorLocalReference.Pieces[KingNum[White]].Alive == false)
			throw runtime_error("Lacks King White on chessboard");

		if (ChessEngineAllPossibleMovesGeneratorLocalReference.Pieces[KingNum[Black]].Alive == false)
			throw runtime_error("Lacks Black White on chessboard");
	}
	CATCH_AND_THROW("finding if there are any kings on chessboard")
}

void ChessEngineDataInitializer::FindStartPositionsOfPieces() 
{
	try
	{
		ClearPiecesData();

		for (PosType CY = 1; CY < MaxChessSizeY; CY++)
			for (PosType CX = 1; CX < MaxChessSizeX; CX++)
			{
				ChessEngineAllPossibleMovesGeneratorLocalReference.ChessBoard[CX][CY] = SpaceNum;

				const bool FoundChessMainPiece = FindChessMainPiece(CX, CY);
				const bool FoundPromotedPawn = FindChessPromotedPiece(CX, CY, FoundChessMainPiece);

				if (ChessEngineAllPossibleMovesGeneratorLocalReference.ChessBoardStartData[CX][CY] != EmptyFieldString && FoundChessMainPiece == false && FoundPromotedPawn == false)
					throw runtime_error(string(string("Bad string ") + ChessEngineAllPossibleMovesGeneratorLocalReference.ChessBoardStartData[CX][CY] + " on chessboard in field(x,y) = (" + to_string(CX) + "," + to_string(CY) + ")").c_str());
			}

		FindIfThereAreKingsOnChessBoard();
	}
	CATCH_AND_THROW("finding start positions of pieces")
}

void ChessEngineDataInitializer::PrintActualStateOfPieces() 
{
	try
	{
		if (ChessEngineConfigurationFileReaderWriterObject.PrintActualStartPositionOfPieces == true)
		{
			for (PieceNumType PieceNum = 0; PieceNum < NumberOfPieces; PieceNum++)
				if (ChessEngineAllPossibleMovesGeneratorLocalReference.Pieces[PieceNum].Alive == true)
					LoggersManagerObject.Log(STREAM(ChessEngineConfigurationFileReaderWriterObject.PieceString[PieceNum] << "(x,y) = (" << int(ChessEngineAllPossibleMovesGeneratorLocalReference.Pieces[PieceNum].ActualPosX) << "," << int(ChessEngineAllPossibleMovesGeneratorLocalReference.Pieces[PieceNum].ActualPosY) << ") Alive = " << ChessEngineAllPossibleMovesGeneratorLocalReference.Pieces[PieceNum].Alive << " Promoted Pawn = " << ChessEngineAllPossibleMovesGeneratorLocalReference.Pieces[PieceNum].Promoted));
		}
	}
	CATCH("printing start positions of pieces")
}
