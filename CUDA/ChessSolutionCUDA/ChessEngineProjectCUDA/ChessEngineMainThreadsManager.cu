
#include "DestinationPlatform.h"

#ifdef WINDOWS_PLATFORM
#include <windows.h>
#endif

#ifdef UNIX_PLATFORM
#include <string.h>
#include <unistd.h>
#include <sys/resource.h>
#endif

#include <chrono>
#include <sstream>

#include "StringUtils.h"
#include "DateTimeUtils.h"
#include "ExceptionsMacro.h"

#include "ChessEngineChessboardFileReader.h"
#include "ChessEngineConfigurationFileReaderWriter.h"

#include "ChessEngineMainThreadsManager.h"

using namespace std;

inline void ChessEngineMainThreadsManager::RegisterGenerateAllPossibleMovesForPieceFunctions(const ThreadNumType MaxThreads)
{
	try
	{
		for (auto& GenerateAllPossibleMovesForPiecesObject : GenerateAllPossibleMovesForPiecesFunctors)
		{
			GenerateAllPossibleMovesForPiecesObject[KingNum[White]] = &ChessEngineAllPossibleMovesGenerator::GenerateAllPossibleMovesForKing<White, Black, StartOfBlackPiecesNum, EndOfBlackPiecesNum>;
			GenerateAllPossibleMovesForPiecesObject[QueenNum[White]] = &ChessEngineAllPossibleMovesGenerator::GenerateAllPossibleMovesForQueen<White, Black, StartOfBlackPiecesNum, EndOfBlackPiecesNum>;

			GenerateAllPossibleMovesForPiecesObject[Rook1Num[White]] = &ChessEngineAllPossibleMovesGenerator::GenerateAllPossibleMovesForRook<White, Black, StartOfBlackPiecesNum, EndOfBlackPiecesNum>;
			GenerateAllPossibleMovesForPiecesObject[Rook2Num[White]] = &ChessEngineAllPossibleMovesGenerator::GenerateAllPossibleMovesForRook<White, Black, StartOfBlackPiecesNum, EndOfBlackPiecesNum>;

			GenerateAllPossibleMovesForPiecesObject[Bishop1Num[White]] = &ChessEngineAllPossibleMovesGenerator::GenerateAllPossibleMovesForBishop<White, Black, StartOfBlackPiecesNum, EndOfBlackPiecesNum>;
			GenerateAllPossibleMovesForPiecesObject[Bishop2Num[White]] = &ChessEngineAllPossibleMovesGenerator::GenerateAllPossibleMovesForBishop<White, Black, StartOfBlackPiecesNum, EndOfBlackPiecesNum>;

			GenerateAllPossibleMovesForPiecesObject[Knight1Num[White]] = &ChessEngineAllPossibleMovesGenerator::GenerateAllPossibleMovesForKnight<White, Black, StartOfBlackPiecesNum, EndOfBlackPiecesNum>;
			GenerateAllPossibleMovesForPiecesObject[Knight2Num[White]] = &ChessEngineAllPossibleMovesGenerator::GenerateAllPossibleMovesForKnight<White, Black, StartOfBlackPiecesNum, EndOfBlackPiecesNum>;

			for (PieceNumType Pawn = Pawn1Num[White]; Pawn <= Pawn8Num[White]; Pawn++)
				GenerateAllPossibleMovesForPiecesObject[Pawn] = &ChessEngineAllPossibleMovesGenerator::GenerateAllPossibleMovesForPawn<White, Black, StartOfBlackPiecesNum, EndOfBlackPiecesNum>;
		}
	}
	CATCH("registering generate all possible move for piece functions")
}

inline void ChessEngineMainThreadsManager::PrepareData(const ThreadNumType MaxThreads, const string& ChessBoardFileName, const uint64_t TestId, const string& ActualDateTimeStr)
{
	try
	{
		LoggersManagerObject.InitializeLoggerManagerDataForTask(string("TEST_" + to_string(TestId)).c_str(), "." OS_DIR_SEP, ActualDateTimeStr, true, 0, function<void(const uint64_t& CurrentThreadId, const uint64_t FileNumber, const string& MessageStr)>());

		for (auto& ChessEngineAllPossibleMovesGeneratorObject : ChessEngineAllPossibleMovesGeneratorObjects)
		{
			ChessEngineAllPossibleMovesGeneratorObject.ChessEngineResultsStatisticsObject.ClearAllStatistics();
			ChessEngineChessboardFileReader(ChessBoardFileName, &ChessEngineAllPossibleMovesGeneratorObject.ChessBoardStartData);
			ChessEngineAllPossibleMovesGeneratorObject.ChessEngineDataInitializerObject.ClearDataForAllPossibleMovesGeneratorAlgorithm();
			ChessEngineAllPossibleMovesGeneratorObject.ChessEngineDataInitializerObject.FindStartPositionsOfPieces();
		}

		RegisterGenerateAllPossibleMovesForPieceFunctions(MaxThreads);

		LoggersManagerObject.Log(STREAM("CHESS GAME SITUATION:"));
		ChessEngineAllPossibleMovesGeneratorObjects[0].ChessEngineChessboardPrinterObject.LogPrintActualChessBoardStateToStreams(&LoggersManager::LogWithoutLineInfoOnlyToFiles, 1);
		ChessEngineAllPossibleMovesGeneratorObjects[0].ChessEngineDataInitializerObject.PrintActualStateOfPieces();
		
		LoggersManagerObject.Log(STREAM(""));
	}
	CATCH_AND_THROW("preparing data")
}

inline void ChessEngineMainThreadsManager::GenerateAllPossibleCombinationsOfMoves(const uint64_t TestId)
{
	try
	{
		if (ChessEngineConfigurationFileReaderWriterObject.MultiThreaded == false)
			ChessEngineAllPossibleMovesGeneratorObjects[0].GenerateAllPossibleCombinationsOfMoves<White, Black, StartOfBlackPiecesNum, EndOfBlackPiecesNum>();
		else
		{
			const bool IsCheck = ChessEngineAllPossibleMovesGeneratorObjects[0].VerifyCheckOfKing<White, Black>();

			bool PossibleMove = false;

			ThreadNumType MaxThreadsForGet = 0;

			ChessPiece PiecesLocalCopy[NumberOfPieces];
			memcpy(PiecesLocalCopy, ChessEngineAllPossibleMovesGeneratorObjects[0].Pieces, (EndOfWhitePiecesNum + 1) * sizeof(ChessPiece));

			future<bool> ResultsFromGenerateAllPossibleMovesForThread[MaxNumberOfThreads];

			for (PieceNumType PieceNum = StartOfWhitePiecesNum; PieceNum <= EndOfWhitePiecesNum; PieceNum++)
				if (PiecesLocalCopy[PieceNum].Alive == true)
				{
					ResultsFromGenerateAllPossibleMovesForThread[MaxThreadsForGet] = async(launch::async, GenerateAllPossibleMovesForPiecesFunctors[MaxThreadsForGet][PieceNum], &ChessEngineAllPossibleMovesGeneratorObjects[MaxThreadsForGet], PiecesLocalCopy[PieceNum].ActualPosX, PiecesLocalCopy[PieceNum].ActualPosY);
					MaxThreadsForGet++;
				}

			for (ThreadNumType ThreadForGetPos = 0; ThreadForGetPos < MaxThreadsForGet; ThreadForGetPos++)
				if (ResultsFromGenerateAllPossibleMovesForThread[ThreadForGetPos].get() == true)
					PossibleMove = true;

			ChessEngineAllPossibleMovesGeneratorObjects[0].UpdateDataForIsCheckAndPossibleMove<White, Black, StartOfBlackPiecesNum, EndOfBlackPiecesNum>(IsCheck, PossibleMove);
		}
	}
	CATCH("generating all possible combinations of moves")
}

inline void ChessEngineMainThreadsManager::SumStatisticsFromAllChessEngineAllPossibleMovesGeneratorObjectsThreads(const ThreadNumType MaxThreads)
{
	try
	{
		for (ThreadNumType ThreadPos = 1; ThreadPos <= MaxThreads; ThreadPos++)
			ChessEngineResultsStatistics::SumStatisticsFromTwoChessEngineResultsStatisticsObjects(ChessEngineAllPossibleMovesGeneratorObjects[0].ChessEngineResultsStatisticsObject, ChessEngineAllPossibleMovesGeneratorObjects[ThreadPos].ChessEngineResultsStatisticsObject);
	}
	CATCH("showing results")
}

inline void ChessEngineMainThreadsManager::InitializeLoggerManagerParameters()
{
	try
	{
		using namespace string_utils;

		LoggersManagerObject.InitializeFilesNames({ "AllMessages", "Information", "Important", "Debugs", "Warnings", "Errors", "Exceptions",  "ErrorsAndExceptions", "Critical", "Statistics" });
		LoggersManagerObject.InitializeSelectiveWordsFunctions(
		{ 
          [](const string& s) { return true; },
          [](const string& s) { return upper_case_str_transform(s).find("INFO") != string::npos; },
          [](const string& s) { return upper_case_str_transform(s).find("IMPORTANT") != string::npos; },
          [](const string& s) { return upper_case_str_transform(s).find("DEBUG") != string::npos; },
          [](const string& s) { return upper_case_str_transform(s).find("WARN") != string::npos; },
          [](const string& s) { return upper_case_str_transform(s).find("ERROR") != string::npos; },
          [](const string& s) { return upper_case_str_transform(s).find("EXCEPTION") != string::npos; },
          [](const string& s) { return upper_case_str_transform(s).find("ERRORS") != string::npos || upper_case_str_transform(s).find("EXCEPTION") != string::npos; },
          [](const string& s) { return upper_case_str_transform(s).find("CRITICAL") != string::npos; },
          [](const string& s) { return upper_case_str_transform(s).find("STATISTICS") != string::npos; }
		});

        LoggersManagerObject.InitializePrintingParameters(ChessEngineConfigurationFileReaderWriterObject.PrintLogToConsole,
                                                          ChessEngineConfigurationFileReaderWriterObject.PrintLogToFiles,
                                                          ChessEngineConfigurationFileReaderWriterObject.PrintLogLineNumberToConsole,
                                                          ChessEngineConfigurationFileReaderWriterObject.PrintLogDateTimeToConsole,
                                                          ChessEngineConfigurationFileReaderWriterObject.PrintLogProcessIdToConsole,
                                                          ChessEngineConfigurationFileReaderWriterObject.PrintLogProcessPriorityLevelToConsole,
                                                          ChessEngineConfigurationFileReaderWriterObject.PrintLogThreadIdToConsole,
                                                          ChessEngineConfigurationFileReaderWriterObject.PrintLogLineNumberToFile,
                                                          ChessEngineConfigurationFileReaderWriterObject.PrintLogDateTimeToFile,
                                                          ChessEngineConfigurationFileReaderWriterObject.PrintLogProcessIdToFile,
                                                          ChessEngineConfigurationFileReaderWriterObject.PrintLogProcessPriorityLevelToFile,
                                                          ChessEngineConfigurationFileReaderWriterObject.PrintLogThreadIdToFile,
                                                          ChessEngineConfigurationFileReaderWriterObject.MaximalNumberOfLinesInOneFile);
	}
	CATCH("initializing logger manager parameter")
}

ChessEngineMainThreadsManager::ChessEngineMainThreadsManager() : ChessEngineKernelManagerCUDAObject(*this)
{	
	try
	{
        #ifdef WINDOWS_PLATFORM
		ChessEngineConfigurationFileReaderWriterObject.SetProcessPriorityHighest == true ? SetPriorityClass(GetCurrentProcess(), REALTIME_PRIORITY_CLASS) : 0;
        #endif

		InitializeLoggerManagerParameters();

		const ThreadNumType MaxThreads = MaxNumberOfThreads - 1;
		
        #ifdef WINDOWS_PLATFORM
		const string ActualDateTimeStr = string("Logs." + GetActualDateTimeWindows(".", ".", ".", ".", ".", "."));
        #endif

        #ifdef UNIX_PLATFORM
        const string ActualDateTimeStr = string("Logs." + GetActualDateTimeStandardCPP(".", ".", ".", ".", "."));
        #endif
																														
		for (const auto& ChessTestObject : ChessEngineConfigurationFileReaderWriterObject.ChessTests)
			if(ChessTestObject.ExecuteTestBool == true)
			{
				ChessEngineKernelManagerCUDAObject.GetMemoryForThreadsDataArray();

				ChessEngineConfigurationFileReaderWriterObject.FirstMove = false;

				ChessEngineConfigurationFileReaderWriterObject.MaxDeepLevel = ChessTestObject.MaxDeepLevel;
				ChessEngineConfigurationFileReaderWriterObject.ReverseColorOfPieces = ChessTestObject.ReverseColorOfPieces;

				PrepareData(MaxThreads, ChessTestObject.ChessBoardFileName, ChessTestObject.TestId, ActualDateTimeStr);
				
				const auto start_time = chrono::high_resolution_clock::now();

				GenerateAllPossibleCombinationsOfMoves(ChessTestObject.TestId);

				const auto stop_time = chrono::high_resolution_clock::now();

				PrintDurationTimeAllFormats(start_time, stop_time, "Execution of looking for mates has taken time: ", "executing printing duration_time");

				SumStatisticsFromAllChessEngineAllPossibleMovesGeneratorObjectsThreads(MaxThreads);
				
				ChessEngineKernelManagerCUDAObject.PrepareDataForKernelAndExecuteKernelCUDA();

				ChessEngineKernelManagerCUDAObject.FreeMemoryForThreadsDataArray();

				ChessEngineAllPossibleMovesGeneratorObjects[0].ChessEngineResultsStatisticsObject.PrintResultsAndStatistics();

				ChessEngineAllPossibleMovesGeneratorObjects[0].ChessEngineResultsStatisticsObject.CompareTestStatistics(ChessTestObject);

				ChessEngineAllPossibleMovesGeneratorObjects[0].ChessEngineResultsStatisticsObject.SaveTestStatisticsToFile(ChessTestObject, ChessTestObject.TestId);
				
				LoggersManagerObject.Log(STREAM("END" << EndLineChar << EndLineChar));
			}

		ChessEngineConfigurationFileReaderWriterObject.WaitForKeyOnTheEnd == true ? getchar() : 0;
	}
	CATCH_AND_WORK("executing chess main threaded engine", "", getchar());
}