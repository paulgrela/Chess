
#include <vector>

#include "ChessEngineConstants.h"
#include "ExceptionsMacro.h"
#include "ChessEngineConfigurationFileReaderWriter.h"

using namespace std;

string GetChessMoveAsString(const PosType cx1, const PosType cy1, const PosType cx2, const PosType cy2)
{
	return char('A' + cx1 - 1) + to_string(cy1) + SpaceChar + char('A' + cx2 - 1) + to_string(cy2);
}

string GetChessMovesPathAsString(const vector<ChessMove>& MovesPath)
{
	string MovesPathStr;

	try
	{
		for (const ChessMove& ChessMove : MovesPath)
			MovesPathStr += ChessEngineConfigurationFileReaderWriterObject.PieceString[ChessMove.PieceNum] + "[" + GetChessMoveAsString(ChessMove.PrevPosX, ChessMove.PrevPosY, ChessMove.NextPosX, ChessMove.NextPosY) + "] ";
	}
	CATCH("getting moves path to string")

	return MovesPathStr;
}