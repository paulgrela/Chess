
#include "DestinationPlatform.h"

#include <fstream>

#include "ChessEngineConstants.h"
#include "ExceptionsMacro.h"

#include "ChessEngineConfigurationFileReaderWriter.h"

using namespace std;

ChessEngineConfigurationFileReaderWriter ChessEngineConfigurationFileReaderWriterObject;

void ChessEngineConfigurationFileReaderWriter::ReadChessConfigurationFile(const char* ConfigFileName)
{
	try
	{
		std::ifstream ChessEngineProjectConfigFile(ConfigFileName);

		MultiThreaded = true;
		SetProcessPriorityHighest = true;

		ReadPreviousMovePathsFromFile = true;

		CompareTestStatistics = false;
		WriteTestsStatisticsToFile = false;

        #ifdef WINDOWS_PLATFORM
		Color1 = 10;
		Color2 = 13;
		Color3 = 10;
		Color4 = 13;
        #endif
        #ifdef UNIX_PLATFORM
		Color1 = 92;
		Color2 = 95;
		Color3 = 92;
		Color4 = 95;
		#endif

		PieceString[QueenNum[White]] = "Q ";
		PieceString[Bishop1Num[White]] = "G1";
		PieceString[Bishop2Num[White]] = "G2";
		PieceString[Rook1Num[White]] = "W1";
		PieceString[Rook2Num[White]] = "W2";
		PieceString[Knight1Num[White]] = "S1";
		PieceString[Knight2Num[White]] = "S2";
		PieceString[Pawn1Num[White]] = "P1";
		PieceString[Pawn1Num[White] + 1] = "P2";
		PieceString[Pawn1Num[White] + 2] = "P3";
		PieceString[Pawn1Num[White] + 3] = "P4";
		PieceString[Pawn1Num[White] + 4] = "P5";
		PieceString[Pawn1Num[White] + 5] = "P6";
		PieceString[Pawn1Num[White] + 6] = "P7";
		PieceString[Pawn8Num[White]] = "P8";
		PieceString[KingNum[White]] = "K ";
		PieceString[QueenNum[Black]] = "q ";
		PieceString[Bishop1Num[Black]] = "g1";
		PieceString[Bishop2Num[Black]] = "g2";
		PieceString[Rook1Num[Black]] = "w1";
		PieceString[Rook2Num[Black]] = "w2";
		PieceString[Knight1Num[Black]] = "s1";
		PieceString[Knight2Num[Black]] = "s2";
		PieceString[Pawn1Num[Black]] = "p1";
		PieceString[Pawn1Num[Black] + 1] = "p2";
		PieceString[Pawn1Num[Black] + 2] = "p3";
		PieceString[Pawn1Num[Black] + 3] = "p4";
		PieceString[Pawn1Num[Black] + 4] = "p5";
		PieceString[Pawn1Num[Black] + 5] = "p6";
		PieceString[Pawn1Num[Black] + 6] = "p7";
		PieceString[Pawn8Num[Black]] = "p8";
		PieceString[KingNum[Black]] = "k ";

		PiecePower[QueenNum[White]] = 16;
		PiecePower[Bishop1Num[White]] = 7;
		PiecePower[Bishop2Num[White]] = 7;
		PiecePower[Rook1Num[White]] = 8;
		PiecePower[Rook2Num[White]] = 8;
		PiecePower[Knight1Num[White]] = 5;
		PiecePower[Knight2Num[White]] = 5;
		PiecePower[Pawn1Num[White]] = 1;
		PiecePower[Pawn1Num[White] + 1] = 1;
		PiecePower[Pawn1Num[White] + 2] = 1;
		PiecePower[Pawn1Num[White] + 3] = 1;
		PiecePower[Pawn1Num[White] + 4] = 1;
		PiecePower[Pawn1Num[White] + 5] = 1;
		PiecePower[Pawn1Num[White] + 6] = 1;
		PiecePower[Pawn8Num[White]] = 1;
		PiecePower[KingNum[White]] = 1000;
		PiecePower[QueenNum[Black]] = 16;
		PiecePower[Bishop1Num[Black]] = 7;
		PiecePower[Bishop2Num[Black]] = 7;
		PiecePower[Rook1Num[Black]] = 8;
		PiecePower[Rook2Num[Black]] = 8;
		PiecePower[Knight1Num[Black]] = 5;
		PiecePower[Knight2Num[Black]] = 5;
		PiecePower[Pawn1Num[Black]] = 1;
		PiecePower[Pawn1Num[Black] + 1] = 1;
		PiecePower[Pawn1Num[Black] + 2] = 1;
		PiecePower[Pawn1Num[Black] + 3] = 1;
		PiecePower[Pawn1Num[Black] + 4] = 1;
		PiecePower[Pawn1Num[Black] + 5] = 1;
		PiecePower[Pawn1Num[Black] + 6] = 1;
		PiecePower[Pawn8Num[Black]] = 1;
		PiecePower[KingNum[Black]] = 1000;

		PrintLogToConsole = true;
		PrintLogToFiles = true;

		PrintLogLineNumberToConsole = false;
		PrintLogDateTimeToConsole = false;
		PrintLogProcessIdToConsole = false;
		PrintLogProcessPriorityLevelToConsole = false;
		PrintLogThreadIdToConsole = false;

		PrintLogLineNumberToFile = true;	
		PrintLogDateTimeToFile = true;
		PrintLogProcessIdToFile = false;
		PrintLogProcessPriorityLevelToFile = false;
		PrintLogThreadIdToFile = true;

		MaximalNumberOfLinesInOneFile = 200;

		PrintActualStartPositionOfPieces = true;
		StopWhenMove = false;
		StopWhenMateFound = false;

		PrintActualMove = false;

		PrintActualChessBoardState = true;
		PrintActualStateWhenMateFound = false;
		CleanScreenBeforePrintingActualState = false;

		PrintPossibleMoveFalseState = false;
		StopWhenPossibleMoveFalseState = false;
		PrintDetailedMoveDescription = false;
		PrintGeneralInterMoveInfo = false;
		PrintChessBoardStateForCheck = false;
		PrintWhenRemovingPiece = false;
		StopWhenRemovingPiece = false;

		ChessEngineProjectConfigFile >> PrintTheHighestScoredMovesPathToCaptureWhenPrintingStatistics;
		ChessEngineProjectConfigFile >> PrintTheHighestScoredMovesPathToCaptureOnChessBoardWhenPrintingStatistics;
		ChessEngineProjectConfigFile >> PrintTheHighestScoredMovesPathToMateWhenPrintingStatistics;
		ChessEngineProjectConfigFile >> PrintTheHighestScoredMovesPathToMateOnChessBoardWhenPrintingStatistics;
		ChessEngineProjectConfigFile >> PrintChosenMovesPathWhenPrintingStatistics;
		ChessEngineProjectConfigFile >> PrintChosenMovesPathOnChessBoardWhenPrintingStatistics;
		ChessEngineProjectConfigFile >> PrintEveryPathToMateWhenPrintingStatistics;
		ChessEngineProjectConfigFile >> PrintEveryMoveToMateOnChessBoardWhenPrintingStatistics;
		ChessEngineProjectConfigFile >> StopWhenPrintEveryMoveToMateOnChessBoardWhenPrintingStatistics;

		PrintOnlyIfChosenMovesPathsToMateIsSmallerThanThisNumber = 11;
		PrintLookingForEqualMoveFromChosenMovesPaths = true;
		ChessEngineProjectConfigFile >> PrintToMovesFile;

		ModuloToWriteMove = 10000000;

		ChessEngineProjectConfigFile >> WaitForKeyOnTheEnd;

		ChessEngineTest ChessTestObject;

		ChessTestObject.TestId = 1;
		ChessTestObject.ExecuteTestBool = true;
		ChessEngineProjectConfigFile >> ChessTestObject.ChessBoardFileName;
		ChessTestObject.MaxDeepLevel = 5;
		ChessEngineProjectConfigFile >> ChessTestObject.ReverseColorOfPieces;
		ChessTestObject.WriteTestStatisticsToFile = true;
		ChessTestObject.PrintTestComparisonResult = true;
		ChessTestObject.NumberOfAllMoves = 355386817;
		ChessTestObject.NumberOfAllWhitePiecesMoves = 18057358;
		ChessTestObject.NumberOfAllBlackPiecesMoves = 337329459;
		ChessTestObject.NumberOfAllWhiteMovesWhenCheckedKing = 35530;
		ChessTestObject.NumberOfAllBlackMovesWhenCheckedKing = 3581321;
		ChessTestObject.NumberOfAllOppositePiecesRemovedByMoveOfWhite = 475193;
		ChessTestObject.NumberOfAllOppositePiecesRemovedByMoveOfBlack = 14163507;
		ChessTestObject.NumberOfAllWhiteMovesCancelledBecauseOfDiscoveredPossibleCheckOfKing = 1996240;
		ChessTestObject.NumberOfAllBlackMovesCancelledBecauseOfDiscoveredPossibleCheckOfKing = 131364094;
		ChessTestObject.NumberOfAllFoundMovesPathsToMateWhite = 2;
		ChessTestObject.NumberOfAllFoundMovesPathsToMateBlack = 184586;
		ChessTestObject.NumberOfAllFoundMovesPathsToStalemate = 162;
		ChessTestObject.TheHighestFoundCaptureMovesPathScoreForAdvantageOfWhite = 46;
		ChessTestObject.TheHighestFoundCaptureMovesPathScoreForAdvantageOfBlack = -111;
		ChessTestObject.TheHighestFoundMateMovesPathScoreForAdvantageOfWhite = -40;
		ChessTestObject.TheHighestFoundMateMovesPathScoreForAdvantageOfBlack = 0;

		ChessTests.push_back(ChessTestObject);
	}
	CATCH("chess print configuration constructor")
}

void ChessEngineConfigurationFileReaderWriter::SaveTestStatisticsToFile(const ChessEngineTest& ChessTestObject) const
{
	try
	{
	}
	CATCH("chess print configuration constructor")
}
