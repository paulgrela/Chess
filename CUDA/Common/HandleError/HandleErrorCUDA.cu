
#ifdef WINDOWS_PLATFORM
#include "Windows.h"
#endif

#include "HandleErrorCUDA.h"

#include "stdio.h"

void HandleError(hipError_t err, const char* file, int line)
{
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		hipDeviceReset();
		exit(EXIT_FAILURE);
	}
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

