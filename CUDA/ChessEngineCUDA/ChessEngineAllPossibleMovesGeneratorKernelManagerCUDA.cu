
#include "DestinationPlatform.h"

#include "TerminalColorsUtils.h"

#include "hip/hip_runtime.h"
#include ""

#include "HandleErrorCUDA.h"

#include "DateTimeUtils.h"
#include "ExceptionsMacro.h"

#include "ChessEngineThreadsData.h"

#include "ChessEngineMainThreadsManager.h"

#include "ChessEngineAllPossibleMovesGeneratorKernelCUDA.h"
#include "ChessEngineAllPossibleMovesGeneratorKernelManagerCUDA.h"

using namespace std;

void ChessEngineAllPossibleMovesGeneratorKernelManagerCUDA::PrintSizesOfStartDataforCUDA()
{
	try
	{
		LoggersManagerObject.Log(STREAM("NUMBER OF ELEMENTS OF ARRAY TO SEND TO CUDA = " << ChessEngineThreadsData::ChessEngineThreadsDataArraySize << EndLineChar));
		LoggersManagerObject.Log(STREAM("SIZE OF ARRAY TO SEND TO CUDA = " << ChessEngineThreadsData::ChessEngineThreadsDataArraySize * sizeof(ChessEngineThreadDataCUDA) << EndLineChar));
		LoggersManagerObject.Log(STREAM("NUMBER OF GPU CUDA USED MEMORY = " << ChessEngineThreadsData::ChessEngineThreadsDataArraySize * sizeof(ChessEngineThreadDataCUDA) + 3 * ChessEngineThreadsData::ChessEngineThreadsDataArraySize * sizeof(ChessScoredMovesPathsCUDA) + 2 * ChessEngineThreadsData::ChessEngineThreadsDataArraySize * sizeof(ChessScoredMovesPathCUDA) << EndLineChar));
	}
	CATCH("printing sizes of start data for cuda")
}

#ifdef WINDOWS_PLATFORM
void ChessEngineAllPossibleMovesGeneratorKernelManagerCUDA::PrintDurationTimesOfCUDAKernelAlgorithmExecution(const float msecTotal1, const float msecTotal2, const float msecTotal3, const chrono::time_point<chrono::steady_clock>& start_time, const chrono::time_point<chrono::steady_clock>& stop_time)
#endif
#ifdef UNIX_PLATFORM
void ChessEngineAllPossibleMovesGeneratorKernelManagerCUDA::PrintDurationTimesOfCUDAKernelAlgorithmExecution(const float msecTotal1, const float msecTotal2, const float msecTotal3, const chrono::system_clock::time_point& start_time, const chrono::system_clock::time_point& stop_time)
#endif
{
	try
	{
		LoggersManagerObject.LogInColorTerminal(terminal_colors_utils::blue, STREAM("TimeCudaEventGPU 1 = " << msecTotal1 << " msec"));
		LoggersManagerObject.LogInColorTerminal(terminal_colors_utils::blue, STREAM("TimeCudaEventGPU 2 = " << msecTotal2 << " msec"));
		LoggersManagerObject.LogInColorTerminal(terminal_colors_utils::blue, STREAM("TimeCudaEventGPU 3 = " << msecTotal3 << " msec"));

		PrintDurationTimeAllFormats(start_time, stop_time, "Execution of adding found paths to vectors of mates paths has taken time: ", "executing printing duration_time");
	}
	CATCH("printing duration times of cuda kernel algorithm execution")
}

void ChessEngineAllPossibleMovesGeneratorKernelManagerCUDA::CopyConstantValuesToCUDA()
{
	try
	{
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(ReverseColorOfPiecesCUDA), &ChessEngineConfigurationFileReaderWriterObject.ReverseColorOfPieces, sizeof(BoolTypeCUDA)));

		LengthTypeCUDA MaxDeepLevelLocal[1] = { static_cast<LengthTypeCUDA>(ChessEngineConfigurationFileReaderWriterObject.MaxDeepLevel) };		
		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(MaxDeepLevelConstantCUDA), MaxDeepLevelLocal, sizeof(LengthTypeCUDA)));

		HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(PiecePowerCUDA), ChessEngineConfigurationFileReaderWriterObject.PiecePower, NumberOfPieces * sizeof(PathScoreTypeCUDA)));

		LoggersManagerObject.Log(STREAM("DeepLevel = " << ChessEngineThreadsData::ChessEngineThreadsDataArrayPointer[0].DeepLevel << " MaxDeepLevel = " << MaxDeepLevelLocal[0] << EndLineChar));
	}
	CATCH("copying constant values to cuda")
}

void ChessEngineAllPossibleMovesGeneratorKernelManagerCUDA::GetMemoryForThreadsDataArray()
{
	try
	{
		ChessEngineThreadsData::ChessEngineThreadsDataArraySize = 0;
		HANDLE_ERROR(hipHostAlloc((void**)&ChessEngineThreadsData::ChessEngineThreadsDataArrayPointer, MaxNumberOfChessBoardsForFastExecution * sizeof(ChessEngineThreadDataCUDA), hipHostMallocDefault));
	}
	CATCH("getting memory for threads data array")
}

void ChessEngineAllPossibleMovesGeneratorKernelManagerCUDA::FreeMemoryForThreadsDataArray()
{
	try
	{
		HANDLE_ERROR(hipHostFree(ChessEngineThreadsData::ChessEngineThreadsDataArrayPointer));
	}
	CATCH("freeing memory for threads data array")
}

void ChessEngineAllPossibleMovesGeneratorKernelManagerCUDA::GetMemoryForChessEngineKernelDataCUDA()
{
	try
	{
		HANDLE_ERROR(hipMalloc((void**)&ThreadStartDataCUDAArrayDevicePointer, ChessEngineThreadsData::ChessEngineThreadsDataArraySize * sizeof(ChessEngineThreadDataCUDA)));
		HANDLE_ERROR(hipHostAlloc((void**)&AllFoundMovesPathsToMateWhitePointer, ChessEngineThreadsData::ChessEngineThreadsDataArraySize * sizeof(ChessScoredMovesPathsCUDA), hipHostMallocDefault));
		HANDLE_ERROR(hipHostAlloc((void**)&AllFoundMovesPathsToMateBlackPointer, ChessEngineThreadsData::ChessEngineThreadsDataArraySize * sizeof(ChessScoredMovesPathsCUDA), hipHostMallocDefault));
		HANDLE_ERROR(hipHostAlloc((void**)&AllFoundMovesPathsToStalematePointer, ChessEngineThreadsData::ChessEngineThreadsDataArraySize * sizeof(ChessScoredMovesPathsCUDA), hipHostMallocDefault));
		HANDLE_ERROR(hipHostAlloc((void**)&TheHighestScoredFoundCaptureMovesPathForAdvantageOfWhitePointer, ChessEngineThreadsData::ChessEngineThreadsDataArraySize * sizeof(ChessScoredMovesPathCUDA), hipHostMallocDefault));
		HANDLE_ERROR(hipHostAlloc((void**)&TheHighestScoredFoundCaptureMovesPathForAdvantageOfBlackPointer, ChessEngineThreadsData::ChessEngineThreadsDataArraySize * sizeof(ChessScoredMovesPathCUDA), hipHostMallocDefault));
	}
	CATCH("getting memory for chess state analyzer kernel data CUDA")
}

void ChessEngineAllPossibleMovesGeneratorKernelManagerCUDA::FreeMemoryForChessEngineKernelDataCUDA()
{
	try
	{
		HANDLE_ERROR(hipFree(ThreadStartDataCUDAArrayDevicePointer));
		HANDLE_ERROR(hipHostFree(AllFoundMovesPathsToMateWhitePointer));
		HANDLE_ERROR(hipHostFree(AllFoundMovesPathsToMateBlackPointer));
		HANDLE_ERROR(hipHostFree(AllFoundMovesPathsToStalematePointer));
		HANDLE_ERROR(hipHostFree(TheHighestScoredFoundCaptureMovesPathForAdvantageOfWhitePointer));
		HANDLE_ERROR(hipHostFree(TheHighestScoredFoundCaptureMovesPathForAdvantageOfBlackPointer));
	}
	CATCH("freeing memory for chess state analyzer kernel data CUDA")
}

void ChessEngineAllPossibleMovesGeneratorKernelManagerCUDA::CopyThreadStartDataCUDAArrayToDevice()
{
	try
	{
		for (uint64_t ThreadStartDataCUDAArrayElementIndex = 0; ThreadStartDataCUDAArrayElementIndex < ChessEngineThreadsData::ChessEngineThreadsDataArraySize; ThreadStartDataCUDAArrayElementIndex++)
		{
			ChessEngineThreadsData::ChessEngineThreadsDataArrayPointer[ThreadStartDataCUDAArrayElementIndex].AllFoundMovesPathsToMatePointer[White] = &AllFoundMovesPathsToMateWhitePointer[ThreadStartDataCUDAArrayElementIndex];
			ChessEngineThreadsData::ChessEngineThreadsDataArrayPointer[ThreadStartDataCUDAArrayElementIndex].AllFoundMovesPathsToMatePointer[Black] = &AllFoundMovesPathsToMateBlackPointer[ThreadStartDataCUDAArrayElementIndex];
			ChessEngineThreadsData::ChessEngineThreadsDataArrayPointer[ThreadStartDataCUDAArrayElementIndex].AllFoundMovesPathsToStalematePointer = &AllFoundMovesPathsToStalematePointer[ThreadStartDataCUDAArrayElementIndex];
			ChessEngineThreadsData::ChessEngineThreadsDataArrayPointer[ThreadStartDataCUDAArrayElementIndex].TheHighestScoredFoundCaptureMovesPathsPointer.WhitePath = &TheHighestScoredFoundCaptureMovesPathForAdvantageOfWhitePointer[ThreadStartDataCUDAArrayElementIndex];
			ChessEngineThreadsData::ChessEngineThreadsDataArrayPointer[ThreadStartDataCUDAArrayElementIndex].TheHighestScoredFoundCaptureMovesPathsPointer.BlackPath = &TheHighestScoredFoundCaptureMovesPathForAdvantageOfBlackPointer[ThreadStartDataCUDAArrayElementIndex];
		}
		HANDLE_ERROR(hipMemcpy(ThreadStartDataCUDAArrayDevicePointer, ChessEngineThreadsData::ChessEngineThreadsDataArrayPointer, ChessEngineThreadsData::ChessEngineThreadsDataArraySize * sizeof(ChessEngineThreadDataCUDA), hipMemcpyHostToDevice));
	}
	CATCH("copying thread start data cuda array to device")
}


void ChessEngineAllPossibleMovesGeneratorKernelManagerCUDA::CopyChessScoredFoundCaptureMovesPathsCUDAToStatisticsFoundMovesPaths(ChessScoredMovesPathCUDA* TheHighestScoredFoundCaptureMovesPathPointer, ChessScoredMovesPath& ChessScoredMovesPathObject, bool (*Comparison)(const PathScoreTypeCUDA x, const PathScoreTypeCUDA y))
{
	try
	{
		for (uint64_t ThreadIndex = 0; ThreadIndex < ChessEngineThreadsData::ChessEngineThreadsDataArraySize; ThreadIndex++)
			if (Comparison(TheHighestScoredFoundCaptureMovesPathPointer[ThreadIndex].Score, ChessScoredMovesPathObject.Score))
				CopyChessScoredMovesPathCUDAObjectToChessScoredMovesPathObject(ChessScoredMovesPathObject, TheHighestScoredFoundCaptureMovesPathPointer[ThreadIndex]);
	}
	CATCH("copying chess scored found capture moves paths cuda to statistics found moves paths")
}

void ChessEngineAllPossibleMovesGeneratorKernelManagerCUDA::CopyChessScoredMovesPathsCUDAToStatisticsFoundMovesPaths(ChessScoredMovesPathsCUDA* const AllFoundMovesPathsToMatePointer, std::vector<ChessScoredMovesPath>& AllFoundMovesPaths, ChessScoredMovesPath& ChessScoredMovesPathObject, bool (*Comparison)(const PathScoreTypeCUDA x, const PathScoreTypeCUDA y))
{
	try
	{
		AllFoundMovesPaths.reserve(AllFoundMovesPaths.size() + ChessEngineThreadsData::ChessEngineThreadsDataArraySize);
		for (uint64_t ThreadIndex = 0; ThreadIndex < ChessEngineThreadsData::ChessEngineThreadsDataArraySize; ThreadIndex++)
			for (uint64_t ChessPathIndex = 0; ChessPathIndex < AllFoundMovesPathsToMatePointer[ThreadIndex].NumberOfFoundMovesPaths; ChessPathIndex++)
			{
				AllFoundMovesPaths.emplace_back(ChessScoredMovesPath(AllFoundMovesPathsToMatePointer[ThreadIndex].MovesPaths[ChessPathIndex].Score, AllFoundMovesPathsToMatePointer[ThreadIndex].MovesPaths[ChessPathIndex].TypeSum, AllFoundMovesPathsToMatePointer[ThreadIndex].MovesPaths[ChessPathIndex].MovesPathLength, AllFoundMovesPathsToMatePointer[ThreadIndex].MovesPaths[ChessPathIndex].MovesPath));

				if (Comparison(AllFoundMovesPathsToMatePointer[ThreadIndex].MovesPaths[ChessPathIndex].Score, ChessScoredMovesPathObject.Score) || (ChessScoredMovesPathObject.MovesPath.empty() == true && AllFoundMovesPathsToMatePointer[ThreadIndex].MovesPaths[ChessPathIndex].MovesPathLength > 0))
					CopyChessScoredMovesPathCUDAObjectToChessScoredMovesPathObject(ChessScoredMovesPathObject, AllFoundMovesPathsToMatePointer[ThreadIndex].MovesPaths[ChessPathIndex]);
			}
	}
	CATCH("copying chess scored moves paths cuda to statistics found moves paths")
}

void ChessEngineAllPossibleMovesGeneratorKernelManagerCUDA::CopyChessScoredMovesPathsCUDAToStatisticsMovesPaths(ChessEngineAllPossibleMovesGenerator& ChessEngineAllPossibleMovesGeneratorObject)
{
	try
	{
		CopyChessScoredMovesPathsCUDAToStatisticsFoundMovesPaths(AllFoundMovesPathsToMateWhitePointer, ChessEngineAllPossibleMovesGeneratorObject.ChessEngineResultsStatisticsObject.AllFoundMovesPathsToMate[White], ChessEngineAllPossibleMovesGeneratorObject.ChessEngineResultsStatisticsObject.TheHighestScoredFoundMateMovesPathForAdvantageOf[Black], [](const PathScoreTypeCUDA Score1, const PathScoreTypeCUDA Score2) { return Score1 > Score2; });
		CopyChessScoredMovesPathsCUDAToStatisticsFoundMovesPaths(AllFoundMovesPathsToMateBlackPointer, ChessEngineAllPossibleMovesGeneratorObject.ChessEngineResultsStatisticsObject.AllFoundMovesPathsToMate[Black], ChessEngineAllPossibleMovesGeneratorObject.ChessEngineResultsStatisticsObject.TheHighestScoredFoundMateMovesPathForAdvantageOf[White], [](const PathScoreTypeCUDA Score1, const PathScoreTypeCUDA Score2) { return Score1 < Score2; });
		CopyChessScoredMovesPathsCUDAToStatisticsFoundMovesPaths(AllFoundMovesPathsToStalematePointer, ChessEngineAllPossibleMovesGeneratorObject.ChessEngineResultsStatisticsObject.AllFoundMovesPathsToStalemate, ChessEngineAllPossibleMovesGeneratorObject.ChessEngineResultsStatisticsObject.TheHighestScoredFoundStalemateMovesPath, [](const PathScoreTypeCUDA Score1, const PathScoreTypeCUDA Score2) { return Score1 < Score2; });
		CopyChessScoredFoundCaptureMovesPathsCUDAToStatisticsFoundMovesPaths(TheHighestScoredFoundCaptureMovesPathForAdvantageOfBlackPointer, ChessEngineAllPossibleMovesGeneratorObject.ChessEngineResultsStatisticsObject.TheHighestScoredFoundCaptureMovesPathForAdvantageOf[Black], [](const PathScoreTypeCUDA Score1, const PathScoreTypeCUDA Score2) { return Score1 < Score2; });
		CopyChessScoredFoundCaptureMovesPathsCUDAToStatisticsFoundMovesPaths(TheHighestScoredFoundCaptureMovesPathForAdvantageOfWhitePointer, ChessEngineAllPossibleMovesGeneratorObject.ChessEngineResultsStatisticsObject.TheHighestScoredFoundCaptureMovesPathForAdvantageOf[White], [](const PathScoreTypeCUDA Score1, const PathScoreTypeCUDA Score2) { return Score1 > Score2; });
	}
	CATCH("copying chess scored moves paths cuda to statistics moves paths")
}

void ChessEngineAllPossibleMovesGeneratorKernelManagerCUDA::PrepareDataForKernelAndExecuteKernelCUDA()
{
	try
	{
		if (ChessEngineThreadsData::ChessEngineThreadsDataArraySize > 0)
		{
			PrintSizesOfStartDataforCUDA();

			CopyConstantValuesToCUDA();

			hipEvent_t StartCudaEvent1;
			HANDLE_ERROR(hipEventCreate(&StartCudaEvent1));
			hipEvent_t StopCudaEvent1;
			HANDLE_ERROR(hipEventCreate(&StopCudaEvent1));

			hipEvent_t StartCudaEvent2;
			HANDLE_ERROR(hipEventCreate(&StartCudaEvent2));
			hipEvent_t StopCudaEvent2;
			HANDLE_ERROR(hipEventCreate(&StopCudaEvent2));

			hipEvent_t StartCudaEvent3;
			HANDLE_ERROR(hipEventCreate(&StartCudaEvent3));
			hipEvent_t StopCudaEvent3;
			HANDLE_ERROR(hipEventCreate(&StopCudaEvent3));

			GetMemoryForChessEngineKernelDataCUDA();
			HANDLE_ERROR(hipEventRecord(StartCudaEvent1, NULL));
			//GetMemoryForChessEngineKernelDataCUDA();
			CopyThreadStartDataCUDAArrayToDevice();
			HANDLE_ERROR(hipEventRecord(StopCudaEvent1, NULL));
			HANDLE_ERROR(hipEventSynchronize(StopCudaEvent1));


			dim3 blocks(1024, 1024);
			dim3 threads(16, 16);
			HANDLE_ERROR(hipEventRecord(StartCudaEvent3, NULL));

			ChessEngineAllPossibleMovesGeneratorKernelCUDA<<<blocks, threads>>>(ThreadStartDataCUDAArrayDevicePointer, static_cast<unsigned int>(ChessEngineThreadsData::ChessEngineThreadsDataArraySize));
			
			HANDLE_ERROR(hipEventRecord(StopCudaEvent3, NULL));
			HANDLE_ERROR(hipEventSynchronize(StopCudaEvent3));


			HANDLE_ERROR(hipEventRecord(StartCudaEvent2, NULL));
			const auto start_time_2 = chrono::high_resolution_clock::now();
			CopyChessScoredMovesPathsCUDAToStatisticsMovesPaths(ChessEngineMainThreadsManagerLocalReference.ChessEngineAllPossibleMovesGeneratorObjects[0]);
			const auto stop_time_2 = chrono::high_resolution_clock::now();
			HANDLE_ERROR(hipEventRecord(StopCudaEvent2, NULL));
			HANDLE_ERROR(hipEventSynchronize(StopCudaEvent2));

			float msecTotal1 = 0.0f;
			HANDLE_ERROR(hipEventElapsedTime(&msecTotal1, StartCudaEvent1, StopCudaEvent1));
			float msecTotal2 = 0.0f;
			HANDLE_ERROR(hipEventElapsedTime(&msecTotal2, StartCudaEvent2, StopCudaEvent2));
			float msecTotal3 = 0.0f;
			HANDLE_ERROR(hipEventElapsedTime(&msecTotal3, StartCudaEvent3, StopCudaEvent3));

			FreeMemoryForChessEngineKernelDataCUDA();

			PrintDurationTimesOfCUDAKernelAlgorithmExecution(msecTotal1, msecTotal2, msecTotal3, start_time_2, stop_time_2);
		}
	}
	CATCH("executing cuda kernel algorithm")
}
