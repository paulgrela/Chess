
#include "DestinationPlatform.h"

#ifdef UNIX_PLATFORM
#include <string.h>
#endif

#include <fstream>
#include <sstream>

#include "ExceptionsMacro.h"

#include "Logger.h"
#include "FileUtils.h"
#include "StringUtils.h"
#include "DateTimeUtils.h"

#include "ChessEngineChessMoveBasicOperations.h"
#include "ChessEngineAllPossibleMovesGenerator.h"

#include "ChessEngineResultsStatistics.h"

#include "TerminalColorsUtils.h"

using namespace std;

void ChessEngineResultsStatistics::SumStatisticsFromTwoChessEngineResultsStatisticsObjects(ChessEngineResultsStatistics& ChessEngineResultsStatisticsObjectMain, const ChessEngineResultsStatistics& ChessEngineResultsStatisticsObjectToAdd)
{
	try
	{
		for (LengthType DeepLevel = 0; DeepLevel < ChessEngineConfigurationFileReaderWriterObject.MaxDeepLevel; DeepLevel++)
			ChessEngineResultsStatisticsObjectMain.NumberOfExecutionsOnLevel[DeepLevel] += ChessEngineResultsStatisticsObjectToAdd.NumberOfExecutionsOnLevel[DeepLevel];

		ChessEngineResultsStatisticsObjectMain.NumberOfAllRecursiveMoves += ChessEngineResultsStatisticsObjectToAdd.NumberOfAllRecursiveMoves;

		for (ColorType Color = White; Color <= Black; Color++)
		{
			ChessEngineResultsStatisticsObjectMain.NumberOfAllMovesWhenCheckedKingColor[Color] += ChessEngineResultsStatisticsObjectToAdd.NumberOfAllMovesWhenCheckedKingColor[Color];
			ChessEngineResultsStatisticsObjectMain.NumberOfAllPiecesMovesByColor[Color] += ChessEngineResultsStatisticsObjectToAdd.NumberOfAllPiecesMovesByColor[Color];
			ChessEngineResultsStatisticsObjectMain.NumberOfAllMovesCancelledBecauseOfDiscoveredPossibleCheckOfKingColor[Color] += ChessEngineResultsStatisticsObjectToAdd.NumberOfAllMovesCancelledBecauseOfDiscoveredPossibleCheckOfKingColor[Color];
			ChessEngineResultsStatisticsObjectMain.NumberOfAllOppositePiecesRemovedByMoveOfColor[Color] += ChessEngineResultsStatisticsObjectToAdd.NumberOfAllOppositePiecesRemovedByMoveOfColor[Color];

			if (ChessEngineResultsStatisticsObjectToAdd.AllFoundMovesPathsToMate[Color].size() > 0)
				ChessEngineResultsStatisticsObjectMain.AllFoundMovesPathsToMate[Color].insert(ChessEngineResultsStatisticsObjectMain.AllFoundMovesPathsToMate[Color].end(), ChessEngineResultsStatisticsObjectToAdd.AllFoundMovesPathsToMate[Color].begin(), ChessEngineResultsStatisticsObjectToAdd.AllFoundMovesPathsToMate[Color].end());
		}

		if (ChessEngineResultsStatisticsObjectToAdd.AllFoundMovesPathsToStalemate.size() > 0)
			ChessEngineResultsStatisticsObjectMain.AllFoundMovesPathsToStalemate.insert(ChessEngineResultsStatisticsObjectMain.AllFoundMovesPathsToStalemate.end(), ChessEngineResultsStatisticsObjectToAdd.AllFoundMovesPathsToStalemate.begin(), ChessEngineResultsStatisticsObjectToAdd.AllFoundMovesPathsToStalemate.end());

		if (ChessEngineResultsStatisticsObjectToAdd.TheHighestScoredFoundCaptureMovesPathForAdvantageOf[White].Score >= ChessEngineResultsStatisticsObjectMain.TheHighestScoredFoundCaptureMovesPathForAdvantageOf[White].Score)
			ChessEngineResultsStatisticsObjectMain.TheHighestScoredFoundCaptureMovesPathForAdvantageOf[White] = ChessEngineResultsStatisticsObjectToAdd.TheHighestScoredFoundCaptureMovesPathForAdvantageOf[White];
		if (ChessEngineResultsStatisticsObjectToAdd.TheHighestScoredFoundCaptureMovesPathForAdvantageOf[Black].Score <= ChessEngineResultsStatisticsObjectMain.TheHighestScoredFoundCaptureMovesPathForAdvantageOf[Black].Score)
			ChessEngineResultsStatisticsObjectMain.TheHighestScoredFoundCaptureMovesPathForAdvantageOf[Black] = ChessEngineResultsStatisticsObjectToAdd.TheHighestScoredFoundCaptureMovesPathForAdvantageOf[Black];

		if (ChessEngineResultsStatisticsObjectToAdd.TheHighestScoredFoundMateMovesPathForAdvantageOf[White].Score < ChessEngineResultsStatisticsObjectMain.TheHighestScoredFoundMateMovesPathForAdvantageOf[White].Score || ChessEngineResultsStatisticsObjectMain.TheHighestScoredFoundMateMovesPathForAdvantageOf[White].MovesPath.empty() == true && ChessEngineResultsStatisticsObjectToAdd.TheHighestScoredFoundMateMovesPathForAdvantageOf[White].MovesPath.empty() == false)
			ChessEngineResultsStatisticsObjectMain.TheHighestScoredFoundMateMovesPathForAdvantageOf[White] = ChessEngineResultsStatisticsObjectToAdd.TheHighestScoredFoundMateMovesPathForAdvantageOf[White];
		if (ChessEngineResultsStatisticsObjectToAdd.TheHighestScoredFoundMateMovesPathForAdvantageOf[Black].Score > ChessEngineResultsStatisticsObjectMain.TheHighestScoredFoundMateMovesPathForAdvantageOf[Black].Score || ChessEngineResultsStatisticsObjectMain.TheHighestScoredFoundMateMovesPathForAdvantageOf[Black].MovesPath.empty() == true && ChessEngineResultsStatisticsObjectToAdd.TheHighestScoredFoundMateMovesPathForAdvantageOf[Black].MovesPath.empty() == false)
			ChessEngineResultsStatisticsObjectMain.TheHighestScoredFoundMateMovesPathForAdvantageOf[Black] = ChessEngineResultsStatisticsObjectToAdd.TheHighestScoredFoundMateMovesPathForAdvantageOf[Black];

		if (ChessEngineResultsStatisticsObjectToAdd.TheHighestScoredFoundStalemateMovesPath.Score <= ChessEngineResultsStatisticsObjectMain.TheHighestScoredFoundStalemateMovesPath.Score)
			ChessEngineResultsStatisticsObjectMain.TheHighestScoredFoundStalemateMovesPath = ChessEngineResultsStatisticsObjectToAdd.TheHighestScoredFoundStalemateMovesPath;
	}
	CATCH("summing all statistics from all threads")
}

void ChessEngineResultsStatistics::ClearAllStatistics()
{
	try
	{
		NumberOfAllRecursiveMoves = 0;

		for (LengthType DeepLevel = 0; DeepLevel < ChessEngineConfigurationFileReaderWriterObject.MaxDeepLevel; DeepLevel++)
			NumberOfExecutionsOnLevel[DeepLevel] = 0;

		for (ColorType Color = White; Color <= Black; Color++)
		{
			NumberOfAllMovesWhenCheckedKingColor[Color] = 0;
			NumberOfAllPiecesMovesByColor[Color] = 0;
			NumberOfAllMovesCancelledBecauseOfDiscoveredPossibleCheckOfKingColor[Color] = 0;
			NumberOfAllOppositePiecesRemovedByMoveOfColor[Color] = 0;
			AllFoundMovesPathsToMate[Color].clear();
			TheHighestScoredFoundCaptureMovesPathForAdvantageOf[Color] = { 0, 0, vector<ChessMove>() };
			TheHighestScoredFoundMateMovesPathForAdvantageOf[Color] = { 0, 0, vector<ChessMove>() };
			TheChosenMovesPathForAdvantageOf[Color] = { 0, 0, vector<ChessMove>() };
		}

		TheHighestScoredFoundStalemateMovesPath = { 0, 0, vector<ChessMove>() };
		AllFoundMovesPathsToStalemate.clear();
	}
	CATCH("clearing all statistics")
}

uint64_t ChessEngineResultsStatistics::GetNumberOfAllMovesTotal() const
{
	return NumberOfAllPiecesMovesByColor[White] + NumberOfAllPiecesMovesByColor[Black];
}

void ChessEngineResultsStatistics::PrintGeneralStatistics() const
{
	try
	{		
		LoggersManagerObject.Log(STREAM("Number of all moves: " << GetNumberOfAllMovesTotal() << EndLineChar));

		LoggersManagerObject.Log(STREAM("Number of all recursive moves: " << NumberOfAllRecursiveMoves << EndLineChar));

		LoggersManagerObject.Log(STREAM("Number of all white pieces moves: " << NumberOfAllPiecesMovesByColor[White]));
		LoggersManagerObject.Log(STREAM("Number of all black pieces moves: " << NumberOfAllPiecesMovesByColor[Black] << EndLineChar));

		LoggersManagerObject.Log(STREAM("Number of all moves when king white is checked: " << NumberOfAllMovesWhenCheckedKingColor[White]));
		LoggersManagerObject.Log(STREAM("Number of all moves when king black is checked: " << NumberOfAllMovesWhenCheckedKingColor[Black] << EndLineChar));

		LoggersManagerObject.Log(STREAM("Number of all pieces lost by black and removed by white: " << NumberOfAllOppositePiecesRemovedByMoveOfColor[White]));
		LoggersManagerObject.Log(STREAM("Number of all pieces lost by white and removed by black: " << NumberOfAllOppositePiecesRemovedByMoveOfColor[Black] << EndLineChar));

		LoggersManagerObject.Log(STREAM("Number of all moves white cancelled because of discovered possible check of king white: " << NumberOfAllMovesCancelledBecauseOfDiscoveredPossibleCheckOfKingColor[White]));
		LoggersManagerObject.Log(STREAM("Number of all moves black cancelled because of discovered possible check of king black: " << NumberOfAllMovesCancelledBecauseOfDiscoveredPossibleCheckOfKingColor[Black] << EndLineChar));
	}
	CATCH("printing general statistics")
}

void ChessEngineResultsStatistics::PrintDetailedStatistics() const
{
	try
	{
		for (LengthType DeepLevel = 0; DeepLevel < ChessEngineConfigurationFileReaderWriterObject.MaxDeepLevel; DeepLevel++)
			LoggersManagerObject.Log(STREAM("Number of exeuctions on level[" << DeepLevel <<"] = " << NumberOfExecutionsOnLevel[DeepLevel]));
		LoggersManagerObject.Log(STREAM(""));
	}
	CATCH("printing detailed statistics")
}

void ChessEngineResultsStatistics::PrintEveryMoveOnChessBoard(const vector<ChessMove>& OnePathToMate)
{
	try
	{
		PieceNumType ChessBoardForCopy[MaxChessSizeX][MaxChessSizeY];

		memcpy(ChessBoardForCopy, ChessEngineAllPossibleMovesGeneratorLocalReference.ChessBoard, sizeof(PieceNumType) * MaxChessSizeX * MaxChessSizeY);

		ChessEngineAllPossibleMovesGeneratorLocalReference.ChessEngineChessboardPrinterObject.LogPrintActualChessBoardStateToStreams(&LoggersManager::LogWithoutLineInfoOnlyToFiles, 0);

		for (const auto& Move : OnePathToMate)
		{
			ChessEngineAllPossibleMovesGeneratorLocalReference.ChessBoard[Move.NextPosX][Move.NextPosY] = ChessEngineAllPossibleMovesGeneratorLocalReference.ChessBoard[Move.PrevPosX][Move.PrevPosY];
			ChessEngineAllPossibleMovesGeneratorLocalReference.ChessBoard[Move.PrevPosX][Move.PrevPosY] = SpaceNum;

			ChessEngineAllPossibleMovesGeneratorLocalReference.ChessEngineChessboardPrinterObject.LogPrintActualChessBoardStateToStreams(&LoggersManager::LogWithoutLineInfoOnlyToFiles, 0);
		}

		memcpy(ChessEngineAllPossibleMovesGeneratorLocalReference.ChessBoard, ChessBoardForCopy, sizeof(PieceNumType) * MaxChessSizeX * MaxChessSizeY);
	}
	CATCH("printing every move to mate on chess board")
}

void ChessEngineResultsStatistics::PrintAllFoundMovesToMatePaths(const PosType Color)
{
	try
	{
		for (auto& OnePathMovesToMate : AllFoundMovesPathsToMate[Color])
		{
			if (ChessEngineConfigurationFileReaderWriterObject.PrintEveryMoveToMateOnChessBoardWhenPrintingStatistics == true)
			{
				LoggersManagerObject.Log(STREAM((EndLineChar)));
				PrintEveryMoveOnChessBoard(OnePathMovesToMate.MovesPath);
			}

			LoggersManagerObject.LogWithoutLineInfo((STREAM("PATH TO MATE " << ColorName[Color] << ": " << GetChessMovesPathAsString(OnePathMovesToMate.MovesPath))));

			if (ChessEngineConfigurationFileReaderWriterObject.PrintEveryMoveToMateOnChessBoardWhenPrintingStatistics == true)
				LoggersManagerObject.LogWithoutLineInfo((STREAM(EndLineChar)));

			if (ChessEngineConfigurationFileReaderWriterObject.StopWhenPrintEveryMoveToMateOnChessBoardWhenPrintingStatistics == true)
				getchar();
		}
	}
	CATCH("printing all found move to mate paths")
}

void ChessEngineResultsStatistics::PrintMovesPath(const char* TaskStr, const PosType Color1, const PosType Color2, const ChessScoredMovesPath& ChessScoredMovesPathObject, const bool PrintTheHighestScoredMovesPathWhenPrintingStatistics, const bool PrintTheHighestScoredMovesPathOnChessBoardWhenPrintingStatistics)
{
	try
	{
		LoggersManagerObject.Log(STREAM(""));

		LoggersManagerObject.Log(STREAM("Score of found best path to " << TaskStr << " " << ColorName[Color2] << "s so " << ColorName[Color1] << "s HAVE ADVANTAGE: Score = " << ChessScoredMovesPathObject.Score << " TypeSum = " << ChessScoredMovesPathObject.TypeSum));
		LoggersManagerObject.Log(STREAM(""));

		if (PrintTheHighestScoredMovesPathOnChessBoardWhenPrintingStatistics == true)
			PrintEveryMoveOnChessBoard(ChessScoredMovesPathObject.MovesPath);

		if (PrintTheHighestScoredMovesPathWhenPrintingStatistics == true)
		{
			LoggersManagerObject.LogWithoutLineInfo((STREAM("PATH TO " << TaskStr << " " << ColorName[Color2] << "s : " << GetChessMovesPathAsString(ChessScoredMovesPathObject.MovesPath))));
			LoggersManagerObject.Log(STREAM(""));
		}
	}
	CATCH("printing single paths")
}

void ChessEngineResultsStatistics::PrintAllFoundMovesPaths()
{
	try
	{
		PrintMovesPath("capture", Black, White, TheHighestScoredFoundCaptureMovesPathForAdvantageOf[Black], ChessEngineConfigurationFileReaderWriterObject.PrintTheHighestScoredMovesPathToCaptureWhenPrintingStatistics, ChessEngineConfigurationFileReaderWriterObject.PrintTheHighestScoredMovesPathToCaptureOnChessBoardWhenPrintingStatistics);
		PrintMovesPath("capture", White, Black, TheHighestScoredFoundCaptureMovesPathForAdvantageOf[White], ChessEngineConfigurationFileReaderWriterObject.PrintTheHighestScoredMovesPathToCaptureWhenPrintingStatistics, ChessEngineConfigurationFileReaderWriterObject.PrintTheHighestScoredMovesPathToCaptureOnChessBoardWhenPrintingStatistics);

		PrintMovesPath("mate", Black, White, TheHighestScoredFoundMateMovesPathForAdvantageOf[Black], ChessEngineConfigurationFileReaderWriterObject.PrintTheHighestScoredMovesPathToMateWhenPrintingStatistics, ChessEngineConfigurationFileReaderWriterObject.PrintTheHighestScoredMovesPathToMateOnChessBoardWhenPrintingStatistics);
		PrintMovesPath("mate", White, Black, TheHighestScoredFoundMateMovesPathForAdvantageOf[White], ChessEngineConfigurationFileReaderWriterObject.PrintTheHighestScoredMovesPathToMateWhenPrintingStatistics, ChessEngineConfigurationFileReaderWriterObject.PrintTheHighestScoredMovesPathToMateOnChessBoardWhenPrintingStatistics);

		ChessEngineBestMoveFinder(*this).ChooseBestMove();

		PrintMovesPath("choose", Black, White, TheChosenMovesPathForAdvantageOf[Black], ChessEngineConfigurationFileReaderWriterObject.PrintChosenMovesPathWhenPrintingStatistics, ChessEngineConfigurationFileReaderWriterObject.PrintChosenMovesPathOnChessBoardWhenPrintingStatistics);
		PrintMovesPath("choose", White, Black, TheChosenMovesPathForAdvantageOf[White], ChessEngineConfigurationFileReaderWriterObject.PrintChosenMovesPathWhenPrintingStatistics, ChessEngineConfigurationFileReaderWriterObject.PrintChosenMovesPathOnChessBoardWhenPrintingStatistics);

		LoggersManagerObject.Log(STREAM(""));

		LoggersManagerObject.LogInColorTerminal(terminal_colors_utils::blue, STREAM("Number of found mates to mate white so BLACKS ARE WINNERS: " << AllFoundMovesPathsToMate[White].size()));
		if (ChessEngineConfigurationFileReaderWriterObject.PrintEveryPathToMateWhenPrintingStatistics == true)
			PrintAllFoundMovesToMatePaths(White);

		LoggersManagerObject.Log(STREAM(""));

		LoggersManagerObject.LogInColorTerminal(terminal_colors_utils::blue, STREAM("Number of found mates to mate black so WHITES ARE WINNERS: " << AllFoundMovesPathsToMate[Black].size()));
		if (ChessEngineConfigurationFileReaderWriterObject.PrintEveryPathToMateWhenPrintingStatistics == true)
			PrintAllFoundMovesToMatePaths(Black);

		LoggersManagerObject.Log(STREAM(""));

		LoggersManagerObject.LogInColorTerminal(terminal_colors_utils::blue, STREAM("Number of found stalemates: " << AllFoundMovesPathsToStalemate.size()));
		if (ChessEngineConfigurationFileReaderWriterObject.PrintEveryPathToMateWhenPrintingStatistics == true)
			for (const auto& OnePathMovesToStalemate : AllFoundMovesPathsToStalemate)
				LoggersManagerObject.Log(STREAM("PATH TO MATE STALEMATE: " << GetChessMovesPathAsString(OnePathMovesToStalemate.MovesPath) << EndLineChar));
	}
	CATCH("printing all found paths")
}

void ChessEngineResultsStatistics::PrintResultsAndStatistics()
{
	try
	{
		LoggersManagerObject.Log(STREAM("Statistics"));
		LoggersManagerObject.Log(STREAM(""));

		PrintGeneralStatistics();
		PrintDetailedStatistics();
		PrintAllFoundMovesPaths();
	}
	CATCH("printing results and statistics");
}

void ChessEngineResultsStatistics::SaveTestStatisticsToFile(const ChessEngineTest& ChessTestObject, const uint64_t TestId) const
{
	try
	{
		if (ChessEngineConfigurationFileReaderWriterObject.WriteTestsStatisticsToFile == true && ChessTestObject.WriteTestStatisticsToFile == true)
		{
			ChessEngineTest ChessTestObject;

			ChessTestObject.TestId = TestId;

			ChessTestObject.NumberOfAllMoves = GetNumberOfAllMovesTotal();

			ChessTestObject.NumberOfAllWhitePiecesMoves = NumberOfAllPiecesMovesByColor[White];
			ChessTestObject.NumberOfAllBlackPiecesMoves = NumberOfAllPiecesMovesByColor[Black];

			ChessTestObject.NumberOfAllWhiteMovesWhenCheckedKing = NumberOfAllMovesWhenCheckedKingColor[White];
			ChessTestObject.NumberOfAllBlackMovesWhenCheckedKing = NumberOfAllMovesWhenCheckedKingColor[Black];

			ChessTestObject.NumberOfAllOppositePiecesRemovedByMoveOfWhite = NumberOfAllOppositePiecesRemovedByMoveOfColor[White];
			ChessTestObject.NumberOfAllOppositePiecesRemovedByMoveOfBlack = NumberOfAllOppositePiecesRemovedByMoveOfColor[Black];

			ChessTestObject.NumberOfAllWhiteMovesCancelledBecauseOfDiscoveredPossibleCheckOfKing = NumberOfAllMovesCancelledBecauseOfDiscoveredPossibleCheckOfKingColor[White];
			ChessTestObject.NumberOfAllBlackMovesCancelledBecauseOfDiscoveredPossibleCheckOfKing = NumberOfAllMovesCancelledBecauseOfDiscoveredPossibleCheckOfKingColor[Black];

			ChessTestObject.NumberOfAllFoundMovesPathsToMateWhite = AllFoundMovesPathsToMate[White].size();
			ChessTestObject.NumberOfAllFoundMovesPathsToMateBlack = AllFoundMovesPathsToMate[Black].size();

			ChessTestObject.NumberOfAllFoundMovesPathsToStalemate = AllFoundMovesPathsToStalemate.size();

			ChessTestObject.TheHighestFoundCaptureMovesPathScoreForAdvantageOfWhite = TheHighestScoredFoundCaptureMovesPathForAdvantageOf[White].Score;
			ChessTestObject.TheHighestFoundCaptureMovesPathScoreForAdvantageOfBlack = TheHighestScoredFoundCaptureMovesPathForAdvantageOf[Black].Score;

			ChessTestObject.TheHighestFoundMateMovesPathScoreForAdvantageOfWhite = TheHighestScoredFoundMateMovesPathForAdvantageOf[White].Score;
			ChessTestObject.TheHighestFoundMateMovesPathScoreForAdvantageOfBlack = TheHighestScoredFoundMateMovesPathForAdvantageOf[Black].Score;

			ChessEngineConfigurationFileReaderWriterObject.SaveTestStatisticsToFile(ChessTestObject);
		}
	}
	CATCH("writing statistics to file")
}

void ChessEngineResultsStatistics::CompareTestStatisticsSingleData(const int64_t DataFromFile, const int64_t DataFromTest, const char* ErrorText, bool& TestOK) const
{
	try
	{
		if (DataFromFile != DataFromTest)
		{
            LoggersManagerObject.LogInColorTerminal(terminal_colors_utils::red, STREAM(ErrorText << DataFromFile << " != " << DataFromTest));
			TestOK = false;
		}
	}
	CATCH("comparing test data")
}

void ChessEngineResultsStatistics::CompareTestStatistics(const ChessEngineTest& ChessTestObject) const
{
	try
	{
		if (ChessEngineConfigurationFileReaderWriterObject.CompareTestStatistics == true && ChessTestObject.PrintTestComparisonResult == true)
		{
			LoggersManagerObject.Log(STREAM(""));
			LoggersManagerObject.Log(STREAM(""));

			bool TestOK = true;

			CompareTestStatisticsSingleData(ChessTestObject.NumberOfAllMoves, GetNumberOfAllMovesTotal(), "TEST ERROR Number of all moves different then awaited: ", TestOK);
			CompareTestStatisticsSingleData(ChessTestObject.NumberOfAllWhitePiecesMoves, NumberOfAllPiecesMovesByColor[White], "TEST ERROR Number of all white pieces moves different then awaited: ", TestOK);
			CompareTestStatisticsSingleData(ChessTestObject.NumberOfAllBlackPiecesMoves, NumberOfAllPiecesMovesByColor[Black], "TEST ERROR Number of all black pieces moves different then awaited: ", TestOK);
			CompareTestStatisticsSingleData(ChessTestObject.NumberOfAllWhiteMovesWhenCheckedKing, NumberOfAllMovesWhenCheckedKingColor[White], "TEST ERROR Number of all white pieces moves when checked king different then awaited: ", TestOK);
			CompareTestStatisticsSingleData(ChessTestObject.NumberOfAllBlackMovesWhenCheckedKing, NumberOfAllMovesWhenCheckedKingColor[Black], "TEST ERROR Number of all black pieces moves when checked king different then awaited: ", TestOK);
			CompareTestStatisticsSingleData(ChessTestObject.NumberOfAllOppositePiecesRemovedByMoveOfWhite, NumberOfAllOppositePiecesRemovedByMoveOfColor[White], "TEST ERROR Number of all opposite pieces moves removed bt move of white different then awaited: ", TestOK);
			CompareTestStatisticsSingleData(ChessTestObject.NumberOfAllOppositePiecesRemovedByMoveOfBlack, NumberOfAllOppositePiecesRemovedByMoveOfColor[Black], "TEST ERROR Number of all opposite pieces moves removed bt move of black different then awaited: ", TestOK);
			CompareTestStatisticsSingleData(ChessTestObject.NumberOfAllWhiteMovesCancelledBecauseOfDiscoveredPossibleCheckOfKing, NumberOfAllMovesCancelledBecauseOfDiscoveredPossibleCheckOfKingColor[White], "TEST ERROR Number of all white moves cancelled because of discovered possible check of king white different then awaited: ", TestOK);
			CompareTestStatisticsSingleData(ChessTestObject.NumberOfAllBlackMovesCancelledBecauseOfDiscoveredPossibleCheckOfKing, NumberOfAllMovesCancelledBecauseOfDiscoveredPossibleCheckOfKingColor[Black], "TEST ERROR Number of all white moves cancelled because of discovered possible check of king black different then awaited: ", TestOK);
			CompareTestStatisticsSingleData(ChessTestObject.NumberOfAllFoundMovesPathsToMateWhite, AllFoundMovesPathsToMate[White].size(), "TEST ERROR Number of all found moves paths to mate white different then awaited: ", TestOK);			
			CompareTestStatisticsSingleData(ChessTestObject.NumberOfAllFoundMovesPathsToMateBlack, AllFoundMovesPathsToMate[Black].size(), "TEST ERROR Number of all found moves paths to mate black different then awaited: ", TestOK);
			CompareTestStatisticsSingleData(ChessTestObject.NumberOfAllFoundMovesPathsToStalemate, AllFoundMovesPathsToStalemate.size(), "TEST ERROR Number of all found moves paths to stalemate different then awaited: ", TestOK);

			CompareTestStatisticsSingleData(ChessTestObject.TheHighestFoundCaptureMovesPathScoreForAdvantageOfWhite, TheHighestScoredFoundCaptureMovesPathForAdvantageOf[White].Score, "TEST ERROR Score of found best advantage move path to capture black different then awaited: ", TestOK);
			CompareTestStatisticsSingleData(ChessTestObject.TheHighestFoundCaptureMovesPathScoreForAdvantageOfBlack, TheHighestScoredFoundCaptureMovesPathForAdvantageOf[Black].Score, "TEST ERROR Score of found best advantage move path to capture white different then awaited: ", TestOK);
			CompareTestStatisticsSingleData(ChessTestObject.TheHighestFoundMateMovesPathScoreForAdvantageOfWhite, TheHighestScoredFoundMateMovesPathForAdvantageOf[White].Score, "TEST ERROR Score of found best advantage move path to mate black different then awaited: ", TestOK);
			CompareTestStatisticsSingleData(ChessTestObject.TheHighestFoundMateMovesPathScoreForAdvantageOfBlack, TheHighestScoredFoundMateMovesPathForAdvantageOf[Black].Score, "TEST ERROR Score of found best advantage move path to mate white different then awaited: ", TestOK);

			if (TestOK == true)
               LoggersManagerObject.LogInColorTerminal(terminal_colors_utils::green, STREAM(("Test OK")));

			LoggersManagerObject.Log(STREAM(""));
		}
	}
	CATCH("comparing test statistics")
}