
#include <fstream>

#include "ExceptionsMacro.h"

#include "ChessEngineChessboardFileReader.h"

using namespace std;

ChessEngineChessboardFileReader::ChessEngineChessboardFileReader(const string& FileName, string(*ChessBoardStartData)[MaxChessSizeX][MaxChessSizeY])
{
	try
	{ 
		ifstream ChessBoardFile;

		ChessBoardFile.open(FileName);

		if (ChessBoardFile.is_open() == false)
			throw runtime_error("Can not open chess board file");

		string SeparationLine;

		PosType CX = 0;
		PosType CY = 0;

		for (string s; getline(ChessBoardFile, s, ChessBoardFieldSeparatorChar);)
		{
			if (s.empty() == false && s.front() == EndLineChar)
			{
				getline(ChessBoardFile, SeparationLine, ChessBoardFieldSeparatorChar);
				CY++;
				CX = 0;
			}
			else
			if (CX < MaxChessSizeX && CY < MaxChessSizeY)
			{
				(*ChessBoardStartData)[CX][CY] = s;
				CX++;
			}
		}

		ChessBoardFile.close();
	}
	CATCH_AND_THROW("reading start chess board")
}